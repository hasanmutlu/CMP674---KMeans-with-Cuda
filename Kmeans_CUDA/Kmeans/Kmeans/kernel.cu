#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define int_ptr int*
#define double_ptr double*
#define X(point) point[0]
#define Y(point) point[1]

#pragma region Random initializer methods
__global__ void setup_rand_kernel(hiprandState *state)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	hiprand_init(1234, id, 0, &state[id]);
}

__global__ void generate_rand_kernel(hiprandState *state, int n, int_ptr result)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	int x;
	hiprandState localState = state[id];
	x = (hiprand(&localState) % 100 * 2) - 100;
	state[id] = localState;
	result[id] += x;
}

#pragma endregion

//Get distance of point to given Centroid
__device__ double get_distance_to_centroid(double_ptr centroid, int_ptr point)
{
	int point_x = X(point);
	int point_y = Y(point);
	double centroid_x = X(centroid);
	double centroid_y = Y(centroid);
	double diffX = point_x - centroid_x;
	double diffY = point_y - centroid_y;
	return sqrtf(diffX * diffX + diffY * diffY);
}

//Find centroid which distance is minumum to given point 
__device__ int get_min_distance_centroid(double_ptr centroids , int_ptr point, int centroid_count)
{
	int point_x = X(point);
	int point_y = Y(point);
	int min = 0;
	double min_value = 0;
	for (int i=0 ; i< centroid_count; i+=2)
	{
		double distance = get_distance_to_centroid(centroids + i, point);
		if (min == 0 || distance < min_value)
		{
			min = i / 2;
			min_value = distance;
		}
	}
	return min;
}

//recenter given centroid according to points of it
__device__ double_ptr recenter_centroid(double_ptr centroid, int_ptr points, int point_count)
{
	double result[2];
	X(result) = 0;
	Y(result) = 0;
	for (int i=0 ; i<point_count *2 ; i+=2)
	{
		X(result) += points[i] / point_count;
		Y(result) += points[i + 1] / point_count;
	}
	return result;
}

__global__ void kmeans(int_ptr points, int_ptr result )
{
	int point_count = 4096;
	int centroid_count = 5;
	for (int i=0;i<point_count *2 ; i+=2)
	{



	}
}


int *generate_random_points()
{ 
	hiprandState *devStates;
	int *devResults, *hostResults;
	int sampleCount = 4096 * 2;
	hostResults = new int[sampleCount];
	hipMalloc((void **)&devResults, sampleCount * sizeof(int));
	hipMemset(devResults, 0, sampleCount * sizeof(int));
	hipMalloc((void **)&devStates, sampleCount * sizeof(hiprandState));
	setup_rand_kernel <<<64, 64 >>>(devStates);
	generate_rand_kernel <<<64, 64 >>>(devStates, sampleCount, devResults);
	hipMemcpy(hostResults, devResults, sampleCount * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(devStates);
	hipFree(devResults);
	return hostResults;

}

int_ptr kmeans_cuda()
{
	hipDeviceReset();
	int *points, *devPoints, *devResult, *hostResult;
	int centroidCount = 5;
	int pointCount = 4096;
	points = generate_random_points();
	hostResult = new int[centroidCount * 2];
	
	hipMalloc((void**)devPoints, pointCount * 2 * sizeof(int));
	hipMemcpy(devPoints, points, pointCount * 2 * sizeof(int),hipMemcpyHostToDevice);

	hipMalloc((void**)devResult, centroidCount * 2 * sizeof(int));
	hipMemset(devResult, 0 , centroidCount * 2 * sizeof(int));

	//kmeans << <64, 64 >> > (devPoints, devResult);
	hipMemcpy(hostResult, devResult, centroidCount * 2 * sizeof(int),hipMemcpyDeviceToHost);
	hipFree(devPoints);
	hipFree(devResult);
	hipDeviceReset();
	return hostResult;


}

//Birinci Algoritma
//1.random noktalari olustur
//2.random centroid merkezleri olustur
//3.geriye her bir centroid icin olusturulan sayi kadar olacak sekilde centroidlerin noktalarini bul ve geriye dondur
//4.her bir centroid in suanki merkez noktalarini tut
//5.merkez noktalarini guncelle
//6.eger merkez degismisse 3. adima git 
//7.ciktilari ekrana ve dosyaya yazdir
//8.ciktilari python kodu ile ekrana cizdirebilirsin


int main(int argc, char *argv[])
{
	int sampleCount = 5 * 2;
	int_ptr result = kmeans_cuda();
	for (int i = 0; i < sampleCount; i += 2) {
		int &x = X(result);
		int &y = Y(result);
		printf("X: %d , Y: %d\n", x, y);
	}
	system("pause");
	return 1;
}


