#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define int_ptr int*
#define double_ptr double*
#define X(point) point[0] // returns X of given point
#define Y(point) point[1] // returns Y of given point
#define P(list,i) &(list[i]) //returns i. point from given list

#pragma region Random initializer methods
__global__ void setup_rand_kernel(hiprandState *state, int sampleCount)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if (id < sampleCount)
	{
		hiprand_init(sampleCount, id, 0, &state[id]);
	}
}

__global__ void generate_rand_kernel(hiprandState *state, int sampleCount, int_ptr result)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if (id < sampleCount)
	{
		hiprandState localState = state[id];
		int x = (hiprand(&localState) % 100 * 2) - 100;
		state[id] = localState;
		result[id] += x;
	}
}

#pragma endregion

//Get distance of point to given Centroid
__device__ double get_distance_to_centroid(double_ptr centroid, int_ptr point)
{
	int point_x = X(point);
	int point_y = Y(point);
	double centroid_x = X(centroid);
	double centroid_y = Y(centroid);
	double diffX = point_x - centroid_x;
	double diffY = point_y - centroid_y;
	return sqrtf(diffX * diffX + diffY * diffY);
}

//Find centroid which distance is minumum to given point 
__device__ int get_min_distance_centroid(double_ptr centroids , int_ptr point, int centroid_count)
{
	int point_x = X(point);
	int point_y = Y(point);
	int min = 0;
	double min_value = 0;
	for (int i=0 ; i< centroid_count; i+=2)
	{
		double distance = get_distance_to_centroid(centroids + i, point);
		if (min == 0 || distance < min_value)
		{
			min = i / 2;
			min_value = distance;
		}
	}
	return min;
}

//recenter given centroid according to points of it
__device__ double_ptr recenter_centroid(double_ptr centroid, int_ptr points, int point_count)
{
	double result[2];
	X(result) = 0;
	Y(result) = 0;
	for (int i=0 ; i<point_count *2 ; i+=2)
	{
		X(result) += points[i] / point_count;
		Y(result) += points[i + 1] / point_count;
	}
	return result;
}

__global__ void kmeans(int_ptr points, int_ptr result )
{
	int point_count = 4096;
	int centroid_count = 5;
	for (int i=0;i<point_count *2 ; i+=2)
	{



	}
}


//Birinci Algoritma
//1.random noktalari olustur OK
//2.random centroid merkezleri olustur OK
//3.geriye her bir centroid icin olusturulan sayi kadar olacak sekilde centroidlerin noktalarini bul ve geriye dondur
//4.her bir centroid in suanki merkez noktalarini tut
//5.merkez noktalarini guncelle
//6.eger merkez degismisse 3. adima git 
//7.ciktilari ekrana ve dosyaya yazdir
//8.ciktilari python kodu ile ekrana cizdirebilirsin

void cudaGetRandomPoints(int count, int_ptr &result )
{
	hiprandState *devStates;
	int_ptr devResults;
	int sampleCount = count * 2;
	int blockCount = 0;
	if (count % 32 == 0)
	{
		blockCount = sampleCount / 32;
	}
	else
	{
		blockCount = ((int)(sampleCount / 32)) + 1;
	}
	result = new int[sampleCount];
	hipMalloc((void **)&devResults, sampleCount * sizeof(int));
	hipMemset(devResults, 0, sampleCount * sizeof(int));
	hipMalloc((void **)&devStates, sampleCount * sizeof(hiprandState));
	setup_rand_kernel << <blockCount, 32 >> >(devStates, sampleCount);
	generate_rand_kernel << <blockCount, 32 >> >(devStates, sampleCount, devResults);
	hipMemcpy(result, devResults, sampleCount * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(devStates);
	hipFree(devResults);
}

void print_points(int_ptr list, int count) 
{
	for (int i=0;i<count * 2;i+=2)
	{
		int_ptr point = P(list, i);
		int x = X(point);
		int y = Y(point);
		printf("%d -> (%d,%d)\n",i/2,x,y);
	}
}

bool checkCentroidChanged(double_ptr curCentroid, double_ptr prevCentroid)
{
	return X(curCentroid) == X(prevCentroid) && Y(curCentroid) == Y(prevCentroid);
}


void cudaKmeans()
{
	hipDeviceReset();
	int_ptr points, *devPoints, *devResult, *hostResult,*centroids;
	int centroidCount = 5;
	int pointCount = 4096;
	cudaGetRandomPoints(pointCount, points);
	cudaGetRandomPoints(centroidCount, centroids);



	/*
	hostResult = new int[centroidCount * 2];
	
	hipMalloc((void**)devPoints, pointCount * 2 * sizeof(int));
	hipMemcpy(devPoints, points, pointCount * 2 * sizeof(int),hipMemcpyHostToDevice);

	hipMalloc((void**)devResult, centroidCount * 2 * sizeof(int));
	hipMemset(devResult, 0 , centroidCount * 2 * sizeof(int));

	//kmeans << <64, 64 >> > (devPoints, devResult);
	hipMemcpy(hostResult, devResult, centroidCount * 2 * sizeof(int),hipMemcpyDeviceToHost);
	hipFree(devPoints);
	hipFree(devResult);
	hipDeviceReset();
	*/



}


int main(int argc, char *argv[])
{
	cudaKmeans();
	system("pause");
	return 1;
}


